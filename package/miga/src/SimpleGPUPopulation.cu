#include "hip/hip_runtime.h"
#include "SimpleGPUPopulation.hpp"
#include "cuda_tools.cuh"
#include "hipcub/hipcub.hpp"
#include <hiprand/hiprand_kernel.h>
#include <random>

class SimpleGPUPopulation::cudaParams
{
public:
    hiprandState *prng_states;
};

// Kernels associated with SimpleGPUPopulation class
// Rename this namespace when creating a new implementation
namespace simple_kernel {

__global__ void init_prng(hiprandState *states, const unsigned long int seed)
{
    const int t_idx = threadIdx.x;
    hiprand_init(seed, t_idx, 0, &states[t_idx]);
}

__global__ void kill_and_reproduce(index_t *pop_genome, const index_t num_seqs, const index_t kill_start, const index_t repr_start, const index_t repr_end, hiprandState *prng_states)
{
    extern __shared__ int parent_idx[];
    const int t_idx = threadIdx.x;
    const int stride = blockDim.x;
    const int son = kill_start + blockIdx.x;

    if (t_idx == 0)
    {
        const index_t repr_n = repr_end - repr_start;
        // Using integer casts is not a big deal for the current problem
        // and it's faster than ceilf
        parent_idx[0] = hiprand_uniform(prng_states + son) * repr_n + 1;
    }
    __syncthreads();

    index_t *genome = pop_genome + son * num_seqs;
    index_t *parent_genome = pop_genome + parent_idx[0] * num_seqs;

    for (int i = t_idx; i < num_seqs; i += stride)
    {
        genome[i] = parent_genome[i];
    }
}

__global__ void mutate(index_t *pop_genome, const index_t swaps, const index_t num_seqs, const index_t start, hiprandState *prng_states)
{
    const int entity = start + threadIdx.x;
    index_t *genome = pop_genome + entity * num_seqs;
    hiprandState state = prng_states[entity];

    for (int n = 0; n < swaps; ++n) {
        const int i = hiprand_uniform(&state) * num_seqs;
        const int j = hiprand_uniform(&state) * num_seqs;

        if (i == j)
            continue;

        const index_t temp { genome[i] };
        genome[i] = genome[j];
        genome[j] = temp;
    }

    prng_states[entity] = state;
}

__global__ void reorder_genome(const index_t *indices, index_t *genome, const index_t num_seqs, const index_t pop_size)
{
    extern __shared__ index_t old_genome[];
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= num_seqs)
    {
        return;
    }

    for (int entity = 0; entity < pop_size; ++entity)
    {
        old_genome[entity * blockDim.x + threadIdx.x] = genome[entity * num_seqs + idx];
    }
    __syncthreads();

    for (int entity = 0; entity < pop_size; ++entity)
    {
        genome[entity * num_seqs + idx] = old_genome[indices[entity] * blockDim.x + threadIdx.x];
    }
}

__global__ void fitness(const data_t residual, const data_t scale, const index_t num_seqs, const seq_t q, const index_t *pop_genome, const seq_t *seq_a, const seq_t *seq_b, float *fitness, const data_t *site_prob_a, const data_t *site_prob_b, const index_t num_ic_a)
{
    extern __shared__ int32_t pair_count[];
    const index_t entity = blockIdx.x;
    const index_t ic_a = blockIdx.y;
	const index_t ic_b = blockIdx.z;
    const int t_idx = threadIdx.x;
    const int stride = blockDim.x;
    const index_t *genome = pop_genome + entity * num_seqs;
    float coupling { 0 };

    // Zero pair_count
    for (int i = t_idx; i < q * q; i += stride)
    {
        pair_count[i] = 0;
    }
    __syncthreads();

    // Pairs counting
    for (int i = t_idx; i < num_seqs; i += stride)
    {
        const seq_t aa1 = seq_a[ic_a * num_seqs + i];
        const seq_t aa2 = seq_b[ic_b * num_seqs + genome[i]];
        atomicAdd(pair_count + aa1 * q + aa2, 1);

    }
    __syncthreads();

    // Coupling
    for (int i = t_idx; i < q * q; i += stride)
    {
        const seq_t aa1 = i / q;
        const seq_t aa2 = i % q;

        const float aa1_prob = site_prob_a[ic_a * q + aa1];
        const float aa2_prob = site_prob_b[ic_b * q + aa2];

        const float pair_prob { residual + pair_count[i] * scale };

        coupling += pair_prob * logf(pair_prob / (aa1_prob * aa2_prob));
    }

    atomicAdd(fitness + entity, coupling);
}

};

SimpleGPUPopulation::SimpleGPUPopulation()
: _site_prob_a { nullptr },
  _site_prob_b { nullptr },
  _params { new cudaParams },

  // CUDA variables
  d_sort_buffer { nullptr },
  d_genome { nullptr },
  d_indices { nullptr },
  d_indices_sorted { nullptr },
  d_seq_a { nullptr },
  d_seq_b { nullptr },
  d_fitness { nullptr },
  d_fitness_sorted { nullptr },
  d_site_prob_a { nullptr },
  d_site_prob_b { nullptr }
{
}

SimpleGPUPopulation::~SimpleGPUPopulation()
{
    delete[] _site_prob_a;
    delete[] _site_prob_b;
    delete[] _params;

    free_device_memory();
}

void SimpleGPUPopulation::free_device_memory()
{
    cudaErr( hipFree(d_sort_buffer) );
    cudaErr( hipFree(d_genome) );
    cudaErr( hipFree(d_indices) );
    cudaErr( hipFree(d_indices_sorted) );
    cudaErr( hipFree(d_seq_a) );
    cudaErr( hipFree(d_seq_b) );
    cudaErr( hipFree(d_fitness) );
    cudaErr( hipFree(d_fitness_sorted) );
    cudaErr( hipFree(d_site_prob_a) );
    cudaErr( hipFree(d_site_prob_b) );

    d_sort_buffer = nullptr;
    d_genome = nullptr;
    d_indices = nullptr;
    d_indices_sorted = nullptr;
    d_seq_a = nullptr;
    d_seq_b = nullptr;
    d_fitness = nullptr;
    d_fitness_sorted = nullptr;
    d_site_prob_a = nullptr;
    d_site_prob_b = nullptr;
}

void SimpleGPUPopulation::set_q(const seq_t value)
{
    _q = value;
}

void SimpleGPUPopulation::set_lambda(const data_t value)
{
    _lambda = value;
}

void SimpleGPUPopulation::set_threads(const size_t threads)
{
}

void SimpleGPUPopulation::set_msa(const index_t numSeqs, seq_t *seq_a, const index_t ic_a, seq_t *seq_b, const index_t ic_b)
{
    _num_seqs = numSeqs;
    _num_ic_a = ic_a;
    _num_ic_b = ic_b;
    _seq_a = seq_a;
    _seq_b = seq_b;

    update_site_probs();
}

void SimpleGPUPopulation::set_genome(index_t *genome, const index_t pop_size)
{
    _pop_size = pop_size;
    _genome = genome;
}

void SimpleGPUPopulation::set_fitness(data_t *fitness)
{
    _fitness = fitness;
}

void SimpleGPUPopulation::sort(const bool minimize)
{
    if (_pop_size == 0 || _num_seqs == 0)
    {
        return;
    }

    population_fitness();

    cuda::range<<<1, _threads>>>(d_indices, _pop_size, 0);

    if (minimize)
    {
		hipcub::DeviceRadixSort::SortPairs(d_sort_buffer, _sort_bytes, d_fitness, d_fitness_sorted, d_indices, d_indices_sorted, _pop_size);
    }
    else
    {
        hipcub::DeviceRadixSort::SortPairsDescending(d_sort_buffer, _sort_bytes, d_fitness, d_fitness_sorted, d_indices, d_indices_sorted, _pop_size);
    }

    const int block { _threads };
    const dim3 grid((_num_seqs + block - 1) / block, 1, 1);
	const size_t shared = _pop_size * block * sizeof(index_t);

	cuda::copy<<<1, block>>>(d_fitness_sorted, d_fitness, _pop_size);
	simple_kernel::reorder_genome<<<grid, block, shared>>>(d_indices_sorted, d_genome, _num_seqs, _pop_size);
}

void SimpleGPUPopulation::site_prob(const index_t num_ic, const seq_t *msa, data_t *site_prob)
{
    const data_t residual { _lambda / (_lambda * _q + _num_seqs * _q) };
    const data_t scale { data_t(1.0) / (_lambda + _num_seqs) };

    std::fill(site_prob, site_prob + num_ic * _q, 0.0);

    for (index_t ic = 0; ic < num_ic; ++ic)
    {
        for (index_t seq = 0; seq < _num_seqs; ++seq)
        {
            const seq_t aa { msa[ic * _num_seqs + seq] };
            ++site_prob[ic * _q + aa];
        }
    }
    for (index_t ic = 0; ic < num_ic; ++ic)
    {
        for (index_t aa = 0; aa < _q; ++aa)
        {
            const index_t index { ic * _q + aa };
            site_prob[index] *= scale;
            site_prob[index] += residual;
        }
    }
}

void SimpleGPUPopulation::update_site_probs()
{
    delete[] _site_prob_a;
    delete[] _site_prob_b;

    _site_prob_a = new data_t[_num_ic_a * _q];
    _site_prob_b = new data_t[_num_ic_b * _q];

    site_prob(_num_ic_a, _seq_a, _site_prob_a);
    site_prob(_num_ic_b, _seq_b, _site_prob_b);
}

void SimpleGPUPopulation::kill_and_reproduce(const index_t kill_start, const index_t kill_end, const index_t repr_start, const index_t repr_end)
{
    const int grid { kill_end - kill_start };
    const int block { _threads };

    simple_kernel::kill_and_reproduce<<<grid, block, sizeof(int)>>>(d_genome, _num_seqs, kill_start, repr_start, repr_end, _params->prng_states);
}

void SimpleGPUPopulation::mutate(const double ratio, const index_t start, const index_t end)
{
    const index_t swaps = ratio * _num_seqs;

    simple_kernel::mutate<<<1, end - start>>>(d_genome, swaps, _num_seqs, start, _params->prng_states);
}

void SimpleGPUPopulation::initialize()
{
    check_device();
    init_gpu_data();
}

void SimpleGPUPopulation::finalize()
{
    retrieve_data();
}

void SimpleGPUPopulation::check_device()
{
    int device_count { 0 };
    cudaErr( hipGetDeviceCount(&device_count) );

    if (device_count == 0)
    {
        throw std::runtime_error("No CUDA capable device found");
    }

    // We support only one GPU
    hipSetDevice(0);
    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);

    _warp_size = device_props.warpSize;
    _threads = 2 * _warp_size;
}

void SimpleGPUPopulation::init_gpu_data()
{
    cudaErr(
        hipMalloc(&d_fitness, _pop_size * sizeof(float))
    );
    cudaErr(
        hipMalloc(&d_fitness_sorted, _pop_size * sizeof(float))
    );
    cudaErr(
        hipMalloc(&d_indices_sorted, _pop_size * sizeof(bool))
    );
    cudaErr(
        hipMalloc(&d_indices, _pop_size * sizeof(bool))
    );

    cudaErr(
        hipMalloc(&d_genome, _pop_size * _num_seqs * sizeof(index_t))
    );
    cudaErr(
        cuda::memcpyH2D(d_genome, _genome, _pop_size * _num_seqs * sizeof(index_t))
    );

    cudaErr(
        hipMalloc(&d_seq_a, _num_ic_a * _num_seqs * sizeof(seq_t))
    );
    cudaErr(
        cuda::memcpyH2D(d_seq_a, _seq_a, _num_ic_a * _num_seqs * sizeof(seq_t))
    );

    cudaErr(
        hipMalloc(&d_seq_b, _num_ic_b * _num_seqs * sizeof(seq_t))
    );
    cudaErr(
        cuda::memcpyH2D(d_seq_b, _seq_b, _num_ic_b * _num_seqs * sizeof(seq_t))
    );

    cudaErr(
        hipMalloc(&d_site_prob_a, _num_ic_a * _q * sizeof(data_t))
    );
    cudaErr(
        cuda::memcpyH2D(d_site_prob_a, _site_prob_a, _num_ic_a * _q * sizeof(data_t))
    );

    cudaErr(
        hipMalloc(&d_site_prob_b, _num_ic_b * _q * sizeof(data_t))
    );
    cudaErr(
        cuda::memcpyH2D(d_site_prob_b, _site_prob_b, _num_ic_b * _q * sizeof(data_t))
    );

    cudaErr(
        hipMalloc(&_params->prng_states, _pop_size * sizeof(hiprandState))
    );

    // Allocate storage for sorting
    cudaErr(
        hipcub::DeviceRadixSort::SortPairs(nullptr, _sort_bytes, d_fitness, d_fitness_sorted, d_indices, d_indices_sorted, _pop_size)
    );
    cudaErr(
        hipMalloc(&d_sort_buffer, _sort_bytes)
    );

    // Initializing values
    cuda::fill<<<1, _warp_size>>>(d_fitness, _pop_size, 0.0f);

    // Initialize PRNGs
    std::random_device random_source;
    simple_kernel::init_prng<<<1, _pop_size>>>(_params->prng_states, random_source());

}

void SimpleGPUPopulation::retrieve_data()
{
    cudaErr(
        cuda::memcpyD2H(_genome, d_genome, _pop_size * _num_seqs * sizeof(index_t))
    );

    // Copy fitness back correcting data type
    float *temp_fitness = new float[_pop_size];
    cudaErr(
        cuda::memcpyD2H(temp_fitness, d_fitness, _pop_size * sizeof(float))
    );
    std::copy(temp_fitness, temp_fitness + _pop_size, _fitness);
    delete[] temp_fitness;

    free_device_memory();
}


void SimpleGPUPopulation::population_fitness()
{
    const data_t residual { _lambda / (_lambda * _q * _q + _num_seqs * _q * _q) };
    const data_t scale { data_t(1.0) / (_num_seqs + _lambda) };

	const dim3 grid(_pop_size, _num_ic_a, _num_ic_b);
    const dim3 block(4 * _warp_size, 1, 1);
    const size_t shared { _q * _q * sizeof(int32_t) };

    cuda::fill<<<1, _warp_size>>>(d_fitness, _pop_size, 0.0f);

    simple_kernel::fitness<<<grid, block, shared>>>(residual, scale, _num_seqs, _q, d_genome, d_seq_a, d_seq_b, d_fitness, d_site_prob_a, d_site_prob_b, _num_ic_a);
}
